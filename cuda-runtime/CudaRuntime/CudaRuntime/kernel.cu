﻿#pragma once

// CUDA launch params


// utility and system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>

// CUDA libs
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

// function definitions
int addVectors();
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);
float* generateRandomNumbers(int n, unsigned int seed);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void reduce(float* input_data, float* output_data) {
    extern __shared__ float shared_data[];

    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    shared_data[tid] = input_data[i];
    __syncthreads();

    // do reduction in shared mem
    for (unsigned int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            shared_data[tid] += shared_data[tid + s];
        }
        __syncthreads();
    }

    // write result for this block to global mem
    if (tid == 0) output_data[blockIdx.x] = shared_data[0];
}

// default number of random numbers to generate
const int DEFAULT_RANDOM_NUMBERS = 4;

// default seed for random number generator
const unsigned int DEFAULT_SEED = 123;

// main function
int main()
{
    float* arr = new float[DEFAULT_RANDOM_NUMBERS];
    arr = generateRandomNumbers(DEFAULT_RANDOM_NUMBERS, DEFAULT_SEED);

    for (int i = 0; i < DEFAULT_RANDOM_NUMBERS; i++) {
        std::cout << "arr[" << i << "] = " << arr[i] << std::endl;
    }

    int blocks = 1;
    int threads = 256;

    printf("\nLaunching CUDA kernel with %i blocks and %i threads...\n", blocks, threads);
    reduce <<<blocks, threads>>> (arr, arr);
    
    hipDeviceSynchronize();  // Wait for GPU to finish

    // TODO generate visualization of number generation as image
    // maybe add perlin noise generation

    for (int i = 0; i < DEFAULT_RANDOM_NUMBERS; i++) {
        std::cout << "arr[" << i << "] = " << arr[i] << std::endl;
    }

    printf("\nExiting main...");
    return 0;
}

// random number generator function
float* generateRandomNumbers(int n, unsigned int seed) {
    // create async stream for computation
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    // allocate space on GPU
    float* d_Rand;
    hipMalloc((void**)&d_Rand, n * sizeof(float));

    printf("Seeding rng with %i ...\n", seed);
    hiprandGenerator_t prngGPU;
    hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetStream(prngGPU, stream);
    hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed);
    
    // allocate space for results
    float* h_RandGPU;
    hipHostMalloc(&h_RandGPU, n * sizeof(float), hipHostMallocDefault);

    printf("Generating %i random numbers on GPU...\n", DEFAULT_RANDOM_NUMBERS);
    hiprandGenerateUniform(prngGPU, (float*)d_Rand, n);
    
    printf("Reading back the results...\n\n");
    hipMemcpyAsync(h_RandGPU, d_Rand, n * sizeof(float),
        hipMemcpyDeviceToHost, stream);

    return h_RandGPU;
}

// default starter function
int addVectors()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
    return 0;
}

// helper function for using CUDA to add vectors in parallel
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
