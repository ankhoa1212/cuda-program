﻿#pragma once

// CUDA launch params


// utility and system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <iostream>

// CUDA libs
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

// function definitions
int addVectors();
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);
float* generateRandomNumbers(int n, unsigned int seed);

__global__ void addKernel(int* c, const int* a, const int* b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

// default number of random numbers to generate
const int DEFAULT_RANDOM_NUMBERS = 2560000;

// default seed for random number generator
const unsigned int DEFAULT_SEED = 123;

// main function
int main()
{
    generateRandomNumbers(DEFAULT_RANDOM_NUMBERS, DEFAULT_SEED);

    // TODO generate visualization of number generation
    // maybe with image generation or image matching

    return 0;
}

// random number generator function
float* generateRandomNumbers(int n, unsigned int seed) {
    // create async stream for computation
    hipStream_t stream;
    hipStreamCreateWithFlags(&stream, hipStreamNonBlocking);

    // allocate space on GPU
    float* d_Rand;
    hipMalloc((void**)&d_Rand, n * sizeof(float));

    printf("Seeding rng with %i ...\n", seed);
    hiprandGenerator_t prngGPU;
    hiprandCreateGenerator(&prngGPU, HIPRAND_RNG_PSEUDO_MTGP32);
    hiprandSetStream(prngGPU, stream);
    hiprandSetPseudoRandomGeneratorSeed(prngGPU, seed);
    
    // allocate space for results
    float* h_RandGPU;
    hipHostMalloc(&h_RandGPU, n * sizeof(float), hipHostMallocDefault);

    printf("Generating random numbers on GPU...\n");
    hiprandGenerateUniform(prngGPU, (float*)d_Rand, n);
    
    printf("Reading back the results...\n");
    hipMemcpyAsync(h_RandGPU, d_Rand, n * sizeof(float),
        hipMemcpyDeviceToHost, stream);

    return h_RandGPU;
}

// default starter function
int addVectors()
{
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    
    return 0;
}

// helper function for using CUDA to add vectors in parallel
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel <<<1, size >>> (dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);

    return cudaStatus;
}
